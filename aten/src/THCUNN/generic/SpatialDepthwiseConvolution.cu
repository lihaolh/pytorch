
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialDepthwiseConvolution.cu"
#else

void THNN_(SpatialDepthwiseConvolution_updateOutput)(
                  THCState *state,
                  THCTensor *input,
                  THCTensor *output,
                  THCTensor *weight,
                  THCTensor *bias,
                  int kW, int kH,
                  int dW, int dH,
                  int padW, int padH,
                  int dilationW, int dilationH)
{
  THCUNN_assertSameGPU(state, 3, input, output, weight);

  // Only handle 4D Input Tensors for now
  THAssert(THCTensor_(nDimension)(state, input) == 4);
  THAssert(THCTensor_(nDimension)(state, weight) == 4);

  // We assume that the input and weight Tensors are shaped properly by
  // the caller, so we verify that here to some extent

  // Weight Tensor is shape (output_channels, 1, kH, kW)
  THAssert(weight->size[1] == 1);

  // Input Tensor is shape (N, input_channels, H, W)
  // We verify that the # of output_channels is a multiple of input_channels
  THAssert(weight->size[0] % input->size[1] == 0);

  // Bias has same # of channels as output
  if (bias) {
    THAssert(bias->size[0] == weight->size[0]);
  }

  // Following the behvaior of other THCUNN functions, we shape the output
  // Tensor ourselves

  int batchSize = input->size[0];
  int height = input->size[2];
  int width = input->size[3];
  int outputHeight = (height + 2 * padH - (dilationH * (kH - 1) + 1)) / dH + 1;
  int outputWidth = (width + 2 * padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  int outputChannels = weight->size[0];

  THCTensor_(resize4d)(state, output, batchSize, outputChannels, outputHeight, outputWidth);

  THCDeviceTensor<real, 4> dInput = toDeviceTensor<real, 4>(state, input);
  THCDeviceTensor<real, 4> dWeight = toDeviceTensor<real, 4>(state, weight);
  THCDeviceTensor<real, 4> dOutput = toDeviceTensor<real, 4>(state, output);
  THCDeviceTensor<real, 1> dBias;
  if (bias) {
    dBias = toDeviceTensor<real, 1>(state, bias);
  }

  // Kernel currently relies upon all the Tensors to be contiguous
  THAssert(dInput.isContiguous());
  THAssert(dWeight.isContiguous());
  THAssert(dOutput.isContiguous());

  int inputChannels = input->size[1];
  int depthwiseMultiplier = outputChannels / inputChannels;

  // One thread per output value
  int n = THCTensor_(nElement)(state, output);
  int blocks = GET_BLOCKS(n);
  dim3 grid(blocks);
  dim3 block(CUDA_NUM_THREADS);
  if (kW == 3 && kH == 3) {
  spatialDepthwiseConvolutionUpdateOutput<real, accreal, unsigned int, 3><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    dInput, dOutput, dWeight, dBias, bias != NULL, n, outputChannels, depthwiseMultiplier,
    width, height, outputWidth, outputHeight,
    kW, kH, dW, dH, padW, padH, dilationW, dilationH);
  } else if (kW == 1 && kH == 1) {
  spatialDepthwiseConvolutionUpdateOutput<real, accreal, unsigned int, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    dInput, dOutput, dWeight, dBias, bias != NULL, n, outputChannels, depthwiseMultiplier,
    width, height, outputWidth, outputHeight,
    kW, kH, dW, dH, padW, padH, dilationW, dilationH);
  } else {
  spatialDepthwiseConvolutionUpdateOutput<real, accreal, unsigned int, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    dInput, dOutput, dWeight, dBias, bias != NULL, n, outputChannels, depthwiseMultiplier,
    width, height, outputWidth, outputHeight,
    kW, kH, dW, dH, padW, padH, dilationW, dilationH);
  }

  THCudaCheck(cudaGetLastError());
}

void THNN_(SpatialDepthwiseConvolution_updateGradInput)(
                  THCState *state,
                  THCTensor *input,
                  THCTensor *gradOutput,
                  THCTensor *gradInput,
                  THCTensor *weight,
                  int kW, int kH,
                  int dW, int dH,
                  int padW, int padH,
                  int dilationW, int dilationH)
{
  THCUNN_assertSameGPU(state, 3, gradOutput, gradInput, weight);

  // Only handle 4D Input Tensors for now
  THAssert(THCTensor_(nDimension)(state, input) == 4);
  THAssert(THCTensor_(nDimension)(state, weight) == 4);
  THAssert(THCTensor_(nDimension)(state, gradOutput) == 4);

  // Minimal shape checking, as above
  // Same # of elements in batch
  THAssert(input->size[0] == gradOutput->size[0]);
  // Same # of filters as outputChannels
  THAssert(weight->size[0] == gradOutput->size[1]);

  // Resize GradInput
  THCTensor_(resizeAs)(state, gradInput, input);

  int inputChannels = input->size[1];
  int height = input->size[2];
  int width = input->size[3];

  int outputChannels = gradOutput->size[1];
  int outputHeight = gradOutput->size[2];
  int outputWidth = gradOutput->size[3];

  int depthwiseMultiplier = outputChannels / inputChannels;

  THCDeviceTensor<real, 4> dGradOutput = toDeviceTensor<real, 4>(state, gradOutput);
  THCDeviceTensor<real, 4> dGradInput = toDeviceTensor<real, 4>(state, gradInput);
  THCDeviceTensor<real, 4> dWeight = toDeviceTensor<real, 4>(state, weight);

  // Kernel currently relies upon all the Tensors to be contiguous
  THAssert(dGradOutput.isContiguous());
  THAssert(dGradInput.isContiguous());
  THAssert(dWeight.isContiguous());

  // One thread per gradInput value
  int n = THCTensor_(nElement)(state, gradInput);
  int blocks = GET_BLOCKS(n);
  dim3 grid(blocks);
  dim3 block(CUDA_NUM_THREADS);
  if (kW == 3 && kH == 3) 
    if (dW == 1 && dH == 1){
      spatialDepthwiseConvolutionUpdateGradInput<real, accreal, unsigned int, 3, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else if (dW == 2 && dH == 2) {
      spatialDepthwiseConvolutionUpdateGradInput<real, accreal, unsigned int, 3, 2><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else {
      spatialDepthwiseConvolutionUpdateGradInput<real, accreal, unsigned int, 3, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    }
  else if (kW == 1 && kH == 1) 
    if (dW == 1 && dH == 1){
      spatialDepthwiseConvolutionUpdateGradInput<real, accreal, unsigned int, 1, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else if (dW == 2 && dH == 2) {
      spatialDepthwiseConvolutionUpdateGradInput<real, accreal, unsigned int, 1, 2><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else {
      spatialDepthwiseConvolutionUpdateGradInput<real, accreal, unsigned int, 1, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    }
  else  
    if (dW == 1 && dH == 1){
      spatialDepthwiseConvolutionUpdateGradInput<real, accreal, unsigned int, 0, 1><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else if (dW == 2 && dH == 2) {
      spatialDepthwiseConvolutionUpdateGradInput<real, accreal, unsigned int, 0, 2><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    } else {
      spatialDepthwiseConvolutionUpdateGradInput<real, accreal, unsigned int, 0, 0><<<grid, block, 0, THCState_getCurrentStream(state)>>>(
      dGradOutput, dGradInput, dWeight, n, inputChannels, depthwiseMultiplier, outputChannels, width,
      height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);
    }
  

  THCudaCheck(cudaGetLastError());
}

void THNN_(SpatialDepthwiseConvolution_accGradParameters)(
                  THCState *state,
                  THCTensor *input,
                  THCTensor *gradOutput,
                  THCTensor *gradWeight,
                  int kW, int kH,
                  int dW, int dH,
                  int padW, int padH,
                  int dilationW, int dilationH)
{
  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradWeight);

  // Only handle 4D Input Tensors for now
  THAssert(THCTensor_(nDimension)(state, input) == 4);
  THAssert(THCTensor_(nDimension)(state, gradOutput) == 4);
  THAssert(THCTensor_(nDimension)(state, gradWeight) == 4);

  // Minimal shape checking as above
  // Same # of elements in batch
  THAssert(input->size[0] == gradOutput->size[0]);
  // Same # of filters as outputChannels
  THAssert(gradWeight->size[0] == gradOutput->size[1]);

  int batchSize = input->size[0];
  int inputChannels = input->size[1];
  int height = input->size[2];
  int width = input->size[3];

  int outputChannels = gradOutput->size[1];
  int outputHeight = gradOutput->size[2];
  int outputWidth = gradOutput->size[3];

  int depthwiseMultiplier = outputChannels / inputChannels;

  THCDeviceTensor<real, 4> dGradOutput = toDeviceTensor<real, 4>(state, gradOutput);
  THCDeviceTensor<real, 4> dInput = toDeviceTensor<real, 4>(state, input);
  THCDeviceTensor<real, 4> dGradWeight = toDeviceTensor<real, 4>(state, gradWeight);

  // Kernel currently relies upon all the Tensors to be contiguous
  THAssert(dGradOutput.isContiguous());
  THAssert(dInput.isContiguous());
  THAssert(dGradWeight.isContiguous());

  // We parallelize so that each block computes a single value in gradWeight
  int blocks = outputChannels * kH * kW;


  // Make sure we have enough threads to perform the reduction, and use this number
  // to create the shared memory size for the reduction
  dim3 grid(blocks);
  dim3 block(getGradParamsNumThreads(batchSize));
  int smem = block.x * sizeof(accreal);

  spatialDepthwiseConvolutionAccGradParameters<real, accreal, unsigned int><<<grid, block, smem, THCState_getCurrentStream(state)>>>(
      dGradOutput, dInput, dGradWeight, batchSize, inputChannels, outputChannels, depthwiseMultiplier, 
      width, height, outputWidth, outputHeight, kW, kH, dW, dH, padW, padH, dilationW, dilationH);

  THCudaCheck(cudaGetLastError());
}

#endif
